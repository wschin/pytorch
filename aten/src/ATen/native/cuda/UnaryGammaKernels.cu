#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/Math.h>

namespace at { namespace native {

// See note [Jiterator]
const char digamma_name[] = "digamma";
void digamma_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "digamma_cuda", [&]() {
      jitted_gpu_kernel</*name=*/digamma_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, digamma_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "digamma_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_digamma(a);
      });
    });
  #endif // USE_JITERATOR
}

// See note [Jiterator]
const char trigamma_name[] = "trigamma";
void trigamma_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "trigamma_cuda", [&]() {
      jitted_gpu_kernel</*name=*/trigamma_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, trigamma_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "trigamma_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_trigamma(a);
      });
    });
  #endif // USE_JITERATOR
}

// TODO: jiterate the polygamma kernel (requires supporting heterogenous functor args)
void polygamma_kernel_cuda(TensorIteratorBase& iter, int64_t n) {
  if (n == 0) {
    digamma_kernel_cuda(iter);
  } else if (n == 1) {
    trigamma_kernel_cuda(iter);
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "polygamma_cuda", [&]() {
      gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_polygamma<scalar_t, /*is_cuda=*/true>(int(n), a);
      });
    });
  }
}

const char lgamma_name[] = "lgamma_kernel";
void lgamma_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "lgamma_cuda", [&]() {
      jitted_gpu_kernel</*name=*/lgamma_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, lgamma_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "lgamma_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ::lgamma(a);
      });
    });
  #endif
}

REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
REGISTER_DISPATCH(lgamma_stub, &lgamma_kernel_cuda);

}} // namespace at::native
